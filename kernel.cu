#include "hip/hip_runtime.h"
﻿#include<cstdio>
#include ""
#include "hip/hip_runtime.h""
#include<stdio.h>
#include<stdlib.h>//rand()
#include<windows.h>//performance counter
#include<time.h>
#include<WinBase.h>
#include<io.h>
#include<sys/stat.h>
#include<chrono>
#include<cstring>
#include<string.h>

#define WIDTH 4
#define TILE_WIDTH 2
#define BLOCKSIZE (TILE_WIDTH*TILE_WIDTH)
#define GRIDSIZE (2*2)
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

//#if defined(NDEBUG)
//#define CUDA_CHECK(x)	(x)
// code for release mode
//#else
// code for debug mode debug mode is defined(_DEBUG)
//#define CUDA_CHECK(x)	do{\
//	(x);\
//	hipError_t e = hipGetLastError();\
//	if(hipSuccess!=e){\
//		printf("cuda failure \"%s\" at %s:%d\n", \
//			hipGetErrorString(e),\
//			__FILE__, __LINE__);\
//		exit(1);\
//	}\
//	}while(0)
//#endif

//#define CUDA_CHECK()
//	hipError_t e = hipGetLastError();\
//	if(hipSuccess!=e){\
//		printf("cuda failure \"%s\" at %s:%d\n", \
//			hipGetErrorString(e),\
//			__FILE__, __LINE__);\
//		exit(1);\
//	}
// no ; needed after CUDA_CHECK()

#define CUDA_CHECK() do{\
	hipError_t e = hipGetLastError();\
	if(hipSuccess!=e){\
		printf("cuda failure \"%s\" at %s:%d\n", \
			hipGetErrorString(e),\
			__FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)

using namespace std;
using namespace chrono;

void genData(unsigned* pData, int size)
{
	while (--size) {
		*pData++ = (unsigned)(rand() % 10);
	}
}
__global__ void matmul(unsigned* g_C, const unsigned* g_A, const unsigned* g_B, int width)
{
	unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int gy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	unsigned sum = 0;
	printf("%d번째 g_C를 구하는 과정입니다\n", gy * width + gx);
	for (int m = 0; m < TILE_WIDTH; m++) {//TILE_WIDTH = blockDim.y(g_A의 경우)= blockDim.x(g_B의 경우)
		printf("g_A[%4d]*g_B[%4d] = %u * %u = %u더하고\n", gy*width+(m*TILE_WIDTH+tx), (m*TILE_WIDTH+ty)*width+gx, g_A[gy*width+(m*TILE_WIDTH+tx)]*g_B[(m*TILE_WIDTH+ty)*width+gx], g_A[gy * width + (m * TILE_WIDTH + tx)], g_B[(m * TILE_WIDTH + ty) * width + gx]);
		sum += g_A[gy * width + (m * TILE_WIDTH + tx)]*g_B[(m*TILE_WIDTH+ty)*width+gx];

	}
	g_C[gy * width + gx] = sum;
	printf("결과는 g_C[% 4d][% 4d] = % u\n", gy,gx,g_C[gy*width+gx]);
}
int main(void)
{
	unsigned* pA = NULL;
	unsigned* pB = NULL;
	unsigned* pC = NULL;
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	pA = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pB = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pC = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	genData(pA, TOTALSIZE); genData(pB, TOTALSIZE);
	printf("pA = {%u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u}\n", pA[0], pA[1], pA[2], pA[3], pA[4], pA[5], pA[6], pA[7], pA[8], pA[9], pA[10], pA[11], pA[12], pA[13], pA[14], pA[15]);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("cuda failure \"%s\" at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	else {
		printf("cuda success at %s:%d\n", __FILE__, __LINE__);
	}

	printf("pB = {%u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u}\n", pB[0], pB[1], pB[2], pB[3], pB[4], pB[5], pB[6], pB[7], pB[8], pB[9], pB[10], pB[11], pB[12], pB[13], pB[14], pB[15]);
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("cuda failure \"%s\" at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	else {
		printf("cuda success at %s:%d\n", __FILE__, __LINE__);
	}

	unsigned* pADev = 0;
	unsigned* pBDev = 0;
	unsigned* pCDev = 0;

	hipMalloc((void**)&pADev, TOTALSIZE * sizeof(unsigned));
	CUDA_CHECK();
	(hipMalloc((void**)&pBDev, TOTALSIZE * sizeof(unsigned)));
	CUDA_CHECK();
	(hipMalloc((void**)&pCDev, TOTALSIZE * sizeof(unsigned)));
	CUDA_CHECK();

	(hipMemset(pADev, 0, TOTALSIZE*sizeof(unsigned)));
	CUDA_CHECK();
	(hipMemset(pBDev, 0, TOTALSIZE*sizeof(unsigned)));
	CUDA_CHECK();
	(hipMemset(pCDev, 0, TOTALSIZE*sizeof(unsigned)));
	CUDA_CHECK();

	hipMemcpy(pADev, pA, sizeof(unsigned) * TOTALSIZE, hipMemcpyHostToDevice);
	CUDA_CHECK();
	(hipMemcpy(pBDev, pB, sizeof(unsigned) * TOTALSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK();

	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));

	dim3 gridDim(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH, 1);
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
	matmul<<<gridDim,blockDim>>>(pCDev, pADev, pBDev, WIDTH);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	(hipPeekAtLastError());
	CUDA_CHECK();
	(hipMemcpy(pC, pCDev, sizeof(unsigned) * TOTALSIZE, hipMemcpyDeviceToHost));
	CUDA_CHECK();
	for (int row = 0; row < WIDTH; row++) {
		for (int col = 0; col < WIDTH; col++) {
			printf("pCDev[%4d][%4d]=%u\n", row, col, pC[row * WIDTH + col]);
		}
	}
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda failure at \"%s\" \n", hipGetErrorString(err));
	}
	else {
		printf("cuda success at cuda function\n");
	}
	for (int row = 0; row < WIDTH; row++) {
		for (int col = 0; col < WIDTH; col++) {
			printf("pC[%4d][%4d]=%u\n", row, col, pC[row * WIDTH + col]);
		}
	}
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda failure at \"%s\" \n", hipGetErrorString(err));
	}
	else {
		printf("cuda success at memcpyDeviceToHost\n");
	}
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 1000.0 / freq);
	fflush(stdout);
	free(pA);
	free(pB);
	free(pC);
	(hipFree(pADev));
	CUDA_CHECK();
	(hipFree(pBDev));
	CUDA_CHECK();
	(hipFree(pCDev));
	CUDA_CHECK();
	return 0;
}