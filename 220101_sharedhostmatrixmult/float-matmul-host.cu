﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<string.h>
#include<malloc.h>
#include<windows.h>
#include<time.h>
#include<Winbase.h>
#include<cstring>
#include<cstdio>
#include<math.h>
#include<io.h>
#include<fcntl.h>
#include<sys/stat.h>
#include<chrono>
#include<crt/hip/device_functions.h>

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do{\
	(x);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess!=e){\
		printf("cuda failure \"%s\" at %s:%d\n", \
			hipGetErrorString(e),\
			__FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif


using namespace std;
using namespace chrono;
typedef duration<long long, nano> nanoseconds;
typedef duration<long long, micro> microsecons;
typedef duration<long long, milli> milliseconds;

#define WIDTH 1024
#define TILE_WIDTH 32
#define WARPSIZE TILE_WIDTH
#define GRIDSIZE ((WIDTH/TILE_WIDTH)*(WIDTH/TILE_WIDTH))
#define BLOCKSIZE (TILE_WIDTH*TILE_WIDTH)
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

void genData(float* ptr, unsigned int size)
{
	while (size--) {
		*ptr++ = (float)(rand() % 1000) / 1000.0F;
	}
}

void matmulti_host(const float* g_A, const float* g_B, float* g_C, const int width) {
	for (register int gy = 0; gy < width; gy++) {
		for (register int gx = 0; gx < width; gx++) {
			register float sum = 0.0F;
			for (register int k = 0; k < width; k++) {
				sum += g_A[gy * width + k] * g_B[k * width + gx];
			}
			g_C[gy * width + gx] = sum;
		}
	}
}

__host__ int main(void)
{
	//바로 아래 두 줄은 QueryPerformance로 CUDA이벤트 쓸때는 
	//float형으로 아래의 메모리할당을 해주어야 오류가 안 나길래 적었습니다
	float* pSource = NULL;
	float* pResult = NULL;
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	pSource = (float*)malloc(sizeof(float) * TOTALSIZE);
	pResult = (float*)malloc(sizeof(float) * TOTALSIZE);

	//host변수 선언 및 초기화
	float* pA = NULL;
	float* pB = NULL;
	float* pC = NULL;
	pA = (float*)malloc(sizeof(float) * TOTALSIZE);
	pB = (float*)malloc(sizeof(float) * TOTALSIZE);
	pC = (float*)malloc(sizeof(float) * TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	genData(pA, TOTALSIZE);
	genData(pB, TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	//printf("elasped time : %f usec\n", (double)(cntEnd - cntStart) * 1000000.0 / (double)(freq));
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	matmulti_host(pA, pB, pC, WIDTH);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 1000.0 / (double)freq);
	//마지막에 host변수 메모리해제코드
	free(pSource);
	free(pResult);
	free(pA);
	free(pB);
	free(pC);
}